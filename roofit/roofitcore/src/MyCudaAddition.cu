#include "hip/hip_runtime.h"
#include "MyCudaAddition.h"

#include "Riostream.h"
#include "RooRealSumFunc.h"
#include "RooRealSumPdf.h"
#include "RooProduct.h"
#include "RooErrorHandler.h"
#include "RooArgSet.h"
#include "RooNameReg.h"
#include "RooNLLVar.h"
#include "RooNLLVarNew.h"
#include "RooChi2Var.h"
#include "RooMsgService.h"
#include "RooBatchCompute.h"

#include <algorithm>
#include <cmath>


////////////////////////////////////////////////////////////////////////////////
/// Constructor with a single set consisting of RooAbsReal.
/// \param[in] name Name of the PDF
/// \param[in] title Title
/// \param[in] sumSet The value of the function will be the sum of the values in this set

MyCudaAddition::MyCudaAddition(const char* name, const char* title, const RooArgList& sumSet)
  : RooAbsReal(name, title)
  , _set("!set","set of components",this)
{
  for (const auto comp : sumSet) {
    _set.add(*comp) ;
  }

}


////////////////////////////////////////////////////////////////////////////////
/// Copy constructor

MyCudaAddition::MyCudaAddition(const MyCudaAddition& other, const char* name)
    : RooAbsReal(other, name)
    , _set("!set",this,other._set)
{
}

////////////////////////////////////////////////////////////////////////////////
/// Calculate and return current value of self

double MyCudaAddition::evaluate() const
{
  double sum(0);
  const RooArgSet* nset = _set.nset() ;

  for (auto* comp : static_range_cast<RooAbsReal*>(_set)) {
    const double tmp = comp->getVal(nset);
    sum += tmp ;
  }
  return sum ;
}



__global__
void additionKernel(std::size_t n, double const*x, double const*y, double *output)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i < n) output[i] = x[i] + y[i];
}


////////////////////////////////////////////////////////////////////////////////
/// Compute addition of PDFs in batches.
void MyCudaAddition::computeBatch(hipStream_t* stream, double* output, size_t nEvents, RooFit::Detail::DataMap const& dataMap) const
{
   auto xSpan = dataMap.at(&_set[0]);
   auto ySpan = dataMap.at(&_set[1]);

   if(stream) {
       // CUDA
       std::cout << "MyCudaAddition CUDA" << std::endl;
       int threadsPerBlock = 256;
       int blocksInGrid = std::ceil( double(nEvents) / threadsPerBlock );
       additionKernel<<<blocksInGrid, threadsPerBlock, 0, *stream>>>(nEvents, xSpan.data(), ySpan.data(), output);
   } else {
       // CPU
       std::cout << "MyCudaAddition CPU" << std::endl;
       for(std::size_t i = 0; i < nEvents; ++i) {
          output[i] = xSpan[i] + ySpan[i];
       }
   }
}
